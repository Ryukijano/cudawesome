#%%import numpy as np
from timeit import default_timer as timer
from numba import vectorize

@vectorize(["float32(float32, float32)"], target='gpu')
def VectorAdd(a,b,c):
        return a + b
def main():
    N=32000000 #No of elements per Array
    
    A = np.ones(N, dtype=np.float32)
    B = np.ones(N, dtype=np.float32)
    C = np.zeroes(N, dtype=np.float32)
    
    start = timer()
    VectorAdd(A, B, C)
    vectoradd_time = timer() - start
    
    print("C[:5]="+ str(C[:5]))
    print("C[-5:] = " + str(C[-5:]))
    
    print("VectorAdd took %f seconds " % vectoradd_time)
    
if__name__ == '__main__'
main()
# %%

# %%
