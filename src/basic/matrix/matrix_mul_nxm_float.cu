#include "hip/hip_runtime.h"
/*
 * @Name: matrix_mul_nxm_float.cu
 * @Description: Matrix (NxM) Floating-Point Product.
 * Each matrix is viewed as a single block of memory.
 * Blocks and threads are viewed as a 2D grid.
 * Custom matrix dimension and block size.
 *
 * @Author: Giacomo Marciani <gmarciani@acm.org>
 * @Institution: University of Rome Tor Vergata
 *
 * @Usage: matrix_mul_nxm_float matrixDimX1 matrixDimY1 matrixDimX2 matrixDimY2 blockSize
 *
 * Default values:
 *  matrixDimX1: 4096
 *  matrixDimY1: 4096
 *  matrixDimX2: 4096
 *  matrixDimY2: 4096
 *  blockSize: 32
 */

#include <stdio.h>
#include <math.h>
#include "../../common/error.h"
#include "../../common/random.h"
#include "../../common/matrix.h"
#include "../../common/mathutil.h"

#ifdef DOUBLE
#define REAL double
#else
#define REAL float
#endif

#define EPSILON (float)1e-5

__global__ void matrixMul(const REAL *a, const REAL *b, REAL *c, const unsigned int dimX1, const unsigned int dimY1, const unsigned int dimX2) {
  const unsigned int iX = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int iY = blockIdx.y * blockDim.y + threadIdx.y;

  if (iX >= dimX2 || iY >= dimY1) return;

  const unsigned int pos = iY * dimX2 + iX;

  REAL val = 0.0f;
  for (unsigned int k = 0; k < dimX1; k++) {
    val += a[iY * dimX1 + k] * b[k * dimX2 + iX];
  }

  c[pos] = val;
}

__host__ void gpuMatrixMul(const REAL *a, const REAL *b, REAL *c,
    const unsigned int matrixDimX1, const unsigned int matrixDimY1, const unsigned int matrixDimX2, const unsigned int matrixDimY2,
    const dim3 gridDim, const dim3 blockDim) {
  REAL *dev_a, *dev_b, *dev_c; // device copies of a, b, c
  const unsigned int size_a = matrixDimX1 * matrixDimY1 * sizeof(REAL); // bytes for a
  const unsigned int size_b = matrixDimX2 * matrixDimY2 * sizeof(REAL); // bytes for b
  const unsigned int size_c = matrixDimY1 * matrixDimX2 * sizeof(REAL); // bytes for c

  // allocate device copy of a, b, c
  HANDLE_ERROR(hipMalloc((void**)&dev_a, size_a));
  HANDLE_ERROR(hipMalloc((void**)&dev_b, size_b));
  HANDLE_ERROR(hipMalloc((void**)&dev_c, size_c));

  // copy inputs to device
  HANDLE_ERROR(hipMemcpy(dev_a, a, size_a, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, size_b, hipMemcpyHostToDevice));

  // launch mul() kernel
  matrixMul<<< gridDim, blockDim >>>(dev_a, dev_b, dev_c, matrixDimX1, matrixDimY1, matrixDimX2);

  // copy device result back to host copy of c
  HANDLE_ERROR(hipMemcpy(c, dev_c, size_c, hipMemcpyDeviceToHost));

  // free device
  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipFree(dev_b));
  HANDLE_ERROR(hipFree(dev_c));
}

int main(const int argc, const char **argv) {
  REAL *a, *b, *c;         // host copies of a, b, c
  unsigned int size_a, size_b, size_c; // bytes for a, b, c
  unsigned int matrixDimX1, matrixDimY1, matrixDimX2, matrixDimY2; // matrices dimensions
  unsigned int gridSizeX, gridSizeY; // grid size
  unsigned int blockSize; // block size
  hipDeviceProp_t gpuInfo; // gpu properties

  // check arguments
  if (argc < 6) {
    fprintf(stderr, "Usage: %s matrixDimX1 matrixDimY1 matrixDimX2 matrixDimY2 blockSize\n", argv[0]);
    exit(1);
  }

  matrixDimX1 = atoi(argv[1]);
  matrixDimY1 = atoi(argv[2]);
  matrixDimX2 = atoi(argv[3]);
  matrixDimY2 = atoi(argv[4]);
  blockSize = atoi(argv[5]);

  if (matrixDimX1 < 1) {
    fprintf(stderr, "Error: matrixDimX1 expected >= 1, got %d\n", matrixDimX1);
    exit(1);
  }

  if (matrixDimY1 < 1) {
    fprintf(stderr, "Error: matrixDimY1 expected >= 1, got %d\n", matrixDimY1);
    exit(1);
  }

  if (matrixDimX2 < 1) {
    fprintf(stderr, "Error: matrixDimX2 expected >= 1, got %d\n", matrixDimX2);
    exit(1);
  }

  if (matrixDimY2 != matrixDimX1) {
    fprintf(stderr, "Error: matrixDimY2 expected = matrixDimX1 (%d), got %d\n", matrixDimX1, matrixDimY2);
    exit(1);
  }

  if (!IS_POWER_OF_2(blockSize)) {
    fprintf(stderr, "Error: blockSize expected as power of 2, got %d\n", blockSize);
    exit(1);
  }

  // grid settings
  const unsigned int maxDimX = max(matrixDimX1, matrixDimX2);
  gridSizeX = maxDimX / blockSize;
  if (gridSizeX * blockSize < maxDimX) {
     gridSizeX += 1;
  }
  const unsigned int maxDimY = max(matrixDimY1, matrixDimY2);
  gridSizeY = maxDimY / blockSize;
  if (gridSizeY * blockSize < maxDimY) {
     gridSizeY += 1;
  }
  dim3 gridDim(gridSizeX, gridSizeY);
  dim3 blockDim(blockSize, blockSize);

  size_a = matrixDimX1 * matrixDimY1 * sizeof(REAL);
  size_b = matrixDimX2 * matrixDimY2 * sizeof(REAL);
  size_c = matrixDimY1 * matrixDimX2 * sizeof(REAL);

  HANDLE_ERROR(hipGetDeviceProperties(&gpuInfo, 0));

  printf("------------------------------------\n");
  printf("Matrix (NxM) Floating-Point Product\n");
  printf("------------------------------------\n");
  #ifdef DOUBLE
  printf("FP Precision: Double\n");
  #else
  printf("FP Precision: Single\n");
  #endif
  printf("Matrix Dimension (A): (%d, %d)\n", matrixDimX1, matrixDimY1);
  printf("Matrix Dimension (B): (%d, %d)\n", matrixDimX2, matrixDimY2);
  printf("Grid Size: (%d, %d, %d) (max: (%d, %d, %d))\n",
    gridDim.x, gridDim.y, gridDim.z,
    gpuInfo.maxGridSize[0], gpuInfo.maxGridSize[1], gpuInfo.maxGridSize[2]);
  printf("Block Size: (%d, %d, %d) (max: (%d, %d, %d))\n",
    blockDim.x, blockDim.y, blockDim.z,
    gpuInfo.maxThreadsDim[0], gpuInfo.maxThreadsDim[1], gpuInfo.maxThreadsDim[2]);
  printf("-----------------------------------\n");

  // allocate host copy of a, b, c
  HANDLE_NULL(a = (REAL*)malloc(size_a));
  HANDLE_NULL(b = (REAL*)malloc(size_b));
  HANDLE_NULL(c = (REAL*)malloc(size_c));

  // fill a, b with random data
  #ifdef DOUBLE
  random_matrix_double(a, matrixDimX1, matrixDimY1);
  random_matrix_double(b, matrixDimX2, matrixDimY2);
  #else
  random_matrix_float(a, matrixDimX1, matrixDimY1);
  random_matrix_float(b, matrixDimX2, matrixDimY2);
  #endif

  // launch kernel matrixMul()
  gpuMatrixMul(a, b, c, matrixDimX1, matrixDimY1, matrixDimX2, matrixDimY2, gridDim, blockDim);

  // test result
  REAL *expected;
  HANDLE_NULL(expected = (REAL*)malloc(size_c));
  #ifdef DOUBLE
  matrix_mul_double(a, b, expected, matrixDimX1, matrixDimY1, matrixDimX2);
  const bool correct = matrix_equals_err_double(c, expected, matrixDimX2, matrixDimY1, EPSILON);
  #else
  matrix_mul_float(a, b, expected, matrixDimX1, matrixDimY1, matrixDimX2);
  const bool correct = matrix_equals_err_float(c, expected, matrixDimX2, matrixDimY1, EPSILON);
  #endif
  if (!correct) {
    fprintf(stderr, "Error\n");
  } else {
    printf("Correct\n");
  }

  // free host
  free(a);
  free(b);
  free(c);
  free(expected);

  return 0;
}
