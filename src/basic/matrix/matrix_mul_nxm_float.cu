#include "hip/hip_runtime.h"
/*
 * @Name: matrix_mul_nxm_float.cu
 * @Description: Matrix (NxM) Floating-Point Product.
 * Each matrix is viewed as a single block of memory.
 * Blocks and threads are viewed as a 2D grid.
 * Custom matrix dimension and block size.
 *
 * @Author: Giacomo Marciani <gmarciani@acm.org>
 * @Institution: University of Rome Tor Vergata
 *
 * @Usage: matrix_mul_nxm_float matrixDimX1 matrixDimY1 matrixDimX2 matrixDimY2 blockSize
 *
 * Default values:
 *  matrixDimX1: 4096
 *  matrixDimY1: 4096
 *  matrixDimX2: 4096
 *  matrixDimY2: 4096
 *  blockSize: 32
 */

#include <stdio.h>
#include <math.h>
#include "../../common/error.h"
#include "../../common/random.h"
#include "../../common/matrix.h"
#include "../../common/mathutil.h"

#ifdef DOUBLE
#define REAL double
#else
#define REAL float
#endif

#define EPSILON (float)1e-5

__global__ void mul(const REAL *a, const REAL *b, REAL *c, const unsigned int dimX1, const unsigned int dimY1, const unsigned int dimX2) {
  const unsigned int iX = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int iY = blockIdx.y * blockDim.y + threadIdx.y;

  if (iX >= dimX2 || iY >= dimY1) return;

  const unsigned int pos = iY * dimX2 + iX;

  REAL val = 0.0f;
  for (unsigned int k = 0; k < dimX1; k++) {
    val += a[iY * dimX1 + k] * b[k * dimX2 + iX];
  }

  c[pos] = val;
}

int main(const int argc, const char **argv) {
  REAL *a, *b, *c;         // host copies of a, b, c
  REAL *dev_a, *dev_b, *dev_c; // device copies of a, b, c
  unsigned int size_a, size_b, size_c; // bytes for a, b, c
  unsigned int matrixDimX1, matrixDimY1, matrixDimX2, matrixDimY2; // matrices dimensions
  unsigned int gridSizeX, gridSizeY; // grid size
  unsigned int blockSize; // block size
  hipDeviceProp_t gpuInfo; // gpu properties

  // check arguments
  if (argc < 6) {
    fprintf(stderr, "Usage: %s matrixDimX1 matrixDimY1 matrixDimX2 matrixDimY2 blockSize\n", argv[0]);
    exit(1);
  }

  matrixDimX1 = atoi(argv[1]);
  matrixDimY1 = atoi(argv[2]);
  matrixDimX2 = atoi(argv[3]);
  matrixDimY2 = atoi(argv[4]);
  blockSize = atoi(argv[5]);

  if (matrixDimX1 < 1) {
    fprintf(stderr, "Error: matrixDimX1 expected >= 1, got %d\n", matrixDimX1);
    exit(1);
  }

  if (matrixDimY1 < 1) {
    fprintf(stderr, "Error: matrixDimY1 expected >= 1, got %d\n", matrixDimY1);
    exit(1);
  }

  if (matrixDimX2 < 1) {
    fprintf(stderr, "Error: matrixDimX2 expected >= 1, got %d\n", matrixDimX2);
    exit(1);
  }

  if (matrixDimY2 != matrixDimX1) {
    fprintf(stderr, "Error: matrixDimY2 expected = matrixDimX1 (%d), got %d\n", matrixDimX1, matrixDimY2);
    exit(1);
  }

  if (!IS_POWER_OF_2(blockSize)) {
    fprintf(stderr, "Error: blockSize expected as power of 2, got %d\n", blockSize);
    exit(1);
  }

  // grid settings
  const unsigned int maxDimX = max(matrixDimX1, matrixDimX2);
  gridSizeX = maxDimX / blockSize;
  if (gridSizeX * blockSize < maxDimX) {
     gridSizeX += 1;
  }
  const unsigned int maxDimY = max(matrixDimY1, matrixDimY2);
  gridSizeY = maxDimY / blockSize;
  if (gridSizeY * blockSize < maxDimY) {
     gridSizeY += 1;
  }
  dim3 gridDim(gridSizeX, gridSizeY);
  dim3 blockDim(blockSize, blockSize);

  size_a = matrixDimX1 * matrixDimY1 * sizeof(REAL);
  size_b = matrixDimX2 * matrixDimY2 * sizeof(REAL);
  size_c = matrixDimY1 * matrixDimX2 * sizeof(REAL);

  HANDLE_ERROR(hipGetDeviceProperties(&gpuInfo, 0));

  printf("------------------------------------\n");
  printf("Matrix (NxM) Floating-Point Product\n");
  printf("------------------------------------\n");
  #ifdef DOUBLE
  printf("FP Precision: Double\n");
  #else
  printf("FP Precision: Single\n");
  #endif
  printf("Matrix Dimension (A): (%d, %d)\n", matrixDimX1, matrixDimY1);
  printf("Matrix Dimension (B): (%d, %d)\n", matrixDimX2, matrixDimY2);
  printf("Grid Size: (%d, %d, %d) (max: (%d, %d, %d))\n",
    gridDim.x, gridDim.y, gridDim.z,
    gpuInfo.maxGridSize[0], gpuInfo.maxGridSize[1], gpuInfo.maxGridSize[2]);
  printf("Block Size: (%d, %d, %d) (max: (%d, %d, %d))\n",
    blockDim.x, blockDim.y, blockDim.z,
    gpuInfo.maxThreadsDim[0], gpuInfo.maxThreadsDim[1], gpuInfo.maxThreadsDim[2]);
  printf("-----------------------------------\n");

  // allocate host copy of a, b, c
  HANDLE_NULL(a = (REAL*)malloc(size_a));
  HANDLE_NULL(b = (REAL*)malloc(size_b));
  HANDLE_NULL(c = (REAL*)malloc(size_c));

  // allocate device copy of a, b, c
  HANDLE_ERROR(hipMalloc((void**)&dev_a, size_a));
  HANDLE_ERROR(hipMalloc((void**)&dev_b, size_b));
  HANDLE_ERROR(hipMalloc((void**)&dev_c, size_c));

  // fill a, b with random data
  #ifdef DOUBLE
  random_matrix_double(a, matrixDimX1, matrixDimY1);
  random_matrix_double(b, matrixDimX2, matrixDimY2);
  #else
  random_matrix_float(a, matrixDimX1, matrixDimY1);
  random_matrix_float(b, matrixDimX2, matrixDimY2);
  #endif

  // copy inputs to device
  HANDLE_ERROR(hipMemcpy(dev_a, a, size_a, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, size_b, hipMemcpyHostToDevice));

  // launch mul() kernel
  mul<<< gridDim, blockDim >>>(dev_a, dev_b, dev_c, matrixDimX1, matrixDimY1, matrixDimX2);

  // copy device result back to host copy of c
  HANDLE_ERROR(hipMemcpy(c, dev_c, size_c, hipMemcpyDeviceToHost));

  // test result
  REAL *expected;
  HANDLE_NULL(expected = (REAL*)malloc(size_c));
  #ifdef DOUBLE
  matrix_mul_double(a, b, expected, matrixDimX1, matrixDimY1, matrixDimX2);
  const bool correct = matrix_equals_err_double(c, expected, matrixDimX2, matrixDimY1, EPSILON);
  #else
  matrix_mul_float(a, b, expected, matrixDimX1, matrixDimY1, matrixDimX2);
  const bool correct = matrix_equals_err_float(c, expected, matrixDimX2, matrixDimY1, EPSILON);
  #endif
  if (!correct) {
    fprintf(stderr, "Error\n");
  } else {
    printf("Correct\n");
  }

  // free host
  free(a);
  free(b);
  free(c);
  free(expected);

  // free device
  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipFree(dev_b));
  HANDLE_ERROR(hipFree(dev_c));

  return 0;
}
