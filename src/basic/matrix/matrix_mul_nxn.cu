#include "hip/hip_runtime.h"
/*
 * @Name: matrix_mul_nxn.cu
 * @Description: Multiplication of NxN integer matrices.
 * Each matrix is viewed as a single block of memory.
 * Blocks and threads are viewed as a 2D grid.
 * Custom matrix dimension and block size.
 *
 * @Author: Giacomo Marciani <gmarciani@acm.org>
 * @Institution: University of Rome Tor Vergata
 *
 * @Usage: matrix_mul_nxn matrixDim blockSize
 */

#include <stdio.h>
#include <math.h>
#include "../../common/error.h"
#include "../../common/random.h"
#include "../../common/matrix.h"

__global__ void mul(const double *a, const double *b, double *c, const unsigned int dim) {
  const unsigned int iX = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int iY = blockIdx.y * blockDim.y + threadIdx.y;

  if (iX < dim && iY < dim) {
    const unsigned int pos = iY * dim + iX;
    double val = 0;
    for (unsigned int k = 0; k < dim; k++) {
      val += a[iY * dim + k] * b[k * dim + iX];
    }

    c[pos] = val;
  }
}

int main(const int argc, const char **argv) {
  double *a, *b, *c;         // host copies of a, b, c
  double *dev_a, *dev_b, *dev_c; // device copies of a, b, c
  unsigned int size; // bytes for a, b, c
  unsigned int matrixDim; // matrix dimension
  unsigned int gridSize; // grid size
  unsigned int blockSize; // block size

  if (argc < 3) {
    fprintf(stderr, "Usage: %s matrixDim blockSize\n", argv[0]);
    exit(1);
  }

  matrixDim = atoi(argv[1]);
  blockSize = atoi(argv[2]);

  if (matrixDim < 1) {
    fprintf(stderr, "Error: matrixDim expected >= 1, got %d\n", matrixDim);
    exit(1);
  }

  if (blockSize < 1) {
    fprintf(stderr, "Error: blockSize expected >= 1, got %d\n", blockSize);
    exit(1);
  }

  size = matrixDim * matrixDim * sizeof(double);

  // allocate host copy of a, b, c
  HANDLE_NULL(a = (double*)malloc(size));
  HANDLE_NULL(b = (double*)malloc(size));
  HANDLE_NULL(c = (double*)malloc(size));

  // allocate device copy of a, b, c
  HANDLE_ERROR(hipMalloc((void**)&dev_a, size));
  HANDLE_ERROR(hipMalloc((void**)&dev_b, size));
  HANDLE_ERROR(hipMalloc((void**)&dev_c, size));

  // fill a, b with random data
  random_matrix_double(a, matrixDim, matrixDim);
  random_matrix_double(b, matrixDim, matrixDim);

  // copy inputs to device
  HANDLE_ERROR(hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice));

  // grid settings
  dim3 gridDim, blockDim;
  gridSize = matrixDim / blockSize;
  if (gridSize * blockSize < matrixDim) {
     gridSize += 1;
  }
  blockDim.x = blockSize;
  blockDim.y = blockSize;
  gridDim.x = gridSize;
  gridDim.y = gridSize;

  // launch mul() kernel
  mul<<< gridDim, blockDim >>>(dev_a, dev_b, dev_c, matrixDim);

  // copy device result back to host copy of c
  HANDLE_ERROR(hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost));

  // test result
  double *d;
  HANDLE_NULL(d = (double*)malloc(size));
  matrix_mul_double(a, b, d, matrixDim, matrixDim, matrixDim);
  if (!matrix_equals_double(c, d, matrixDim, matrixDim)) {
    fprintf(stderr, "Error\n");
  } else {
    printf("Correct\n");
  }

  // free host
  free(a);
  free(b);
  free(c);
  free(d);

  // free device
  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipFree(dev_b));
  HANDLE_ERROR(hipFree(dev_c));

  return 0;
}
