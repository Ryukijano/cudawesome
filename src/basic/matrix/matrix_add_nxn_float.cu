#include "hip/hip_runtime.h"
/*
 * @Name: matrix_add_nxn_float.cu
 * @Description: Matrix (NxN) Floating-Point Sum
 * Each matrix is viewed as a single block of memory.
 * Blocks and threads are viewed as a 2D grid.
 * Custom matrix dimension and block size.
 *
 * @Author: Giacomo Marciani <gmarciani@acm.org>
 * @Institution: University of Rome Tor Vergata
 *
 * @Usage: matrix_add_nxn_float matrixDim blockSize
 *
 * Default values:
 *  matrixDim: 4096
 *  blockSize: 32
 */

#include <stdio.h>
#include <math.h>
#include "../../common/error.h"
#include "../../common/random.h"
#include "../../common/matrix.h"

#ifdef DOUBLE
#define REAL double
#else
#define REAL float
#endif

__global__ void add(const REAL *a, const REAL *b, REAL *c, const unsigned int dim) {
  const unsigned int iX = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int iY = blockIdx.y * blockDim.y + threadIdx.y;

  if (iX >= dim || iY >= dim) return;

  const unsigned int pos = iY * dim + iX;
  c[pos] = a[pos] + b[pos];
}

int main(const int argc, const char **argv) {
  REAL *a, *b, *c;             // host copies of a, b, c
  REAL *dev_a, *dev_b, *dev_c; // device copies of a, b, c
  unsigned int size; // bytes for a, b, c
  unsigned int matrixDim; // matrix dimension
  unsigned int gridSize; // grid size
  unsigned int blockSize; // block size
  hipDeviceProp_t gpuInfo; // gpu properties

  // check arguments
  if (argc < 3) {
    fprintf(stderr, "Usage: %s matrixDim blockSize\n", argv[0]);
    exit(1);
  }

  matrixDim = atoi(argv[1]);
  blockSize = atoi(argv[2]);

  if (matrixDim < 1) {
    fprintf(stderr, "Error: matrixDim expected >= 1, got %d\n", matrixDim);
    exit(1);
  }

  if (blockSize < 1) {
    fprintf(stderr, "Error: blockSize expected >= 1, got %d\n", blockSize);
    exit(1);
  }

  // grid settings
  gridSize = matrixDim / blockSize;
  if (gridSize * blockSize < matrixDim) {
     gridSize += 1;
  }
  dim3 gridDim(gridSize, gridSize);
  dim3 blockDim(blockSize, blockSize);

  size = matrixDim * matrixDim * sizeof(REAL);

  HANDLE_ERROR(hipGetDeviceProperties(&gpuInfo, 0));

  printf("------------------------------------\n");
  printf("Matrix (NxN) Floating-Point Sum\n");
  printf("------------------------------------\n");
  #ifdef DOUBLE
  printf("FP Precision: Double\n");
  #else
  printf("FP Precision: Single\n");
  #endif
  printf("Matrix Dimension: (%d, %d)\n", matrixDim, matrixDim);
  printf("Grid Size: (%d, %d, %d) (max: (%d, %d, %d))\n",
    gridDim.x, gridDim.y, gridDim.z,
    gpuInfo.maxGridSize[0], gpuInfo.maxGridSize[1], gpuInfo.maxGridSize[2]);
  printf("Block Size: (%d, %d, %d) (max: (%d, %d, %d))\n",
    blockDim.x, blockDim.y, blockDim.z,
    gpuInfo.maxThreadsDim[0], gpuInfo.maxThreadsDim[1], gpuInfo.maxThreadsDim[2]);
  printf("-----------------------------------\n");

  // allocate host copies of a, b, c
  HANDLE_NULL(a = (REAL*)malloc(size));
  HANDLE_NULL(b = (REAL*)malloc(size));
  HANDLE_NULL(c = (REAL*)malloc(size));

  // allocate device copies of a, b, c
  HANDLE_ERROR(hipMalloc((void**)&dev_a, size));
  HANDLE_ERROR(hipMalloc((void**)&dev_b, size));
  HANDLE_ERROR(hipMalloc((void**)&dev_c, size));

  // fill a, b with random data
  #ifdef DOUBLE
  random_matrix_double(a, matrixDim, matrixDim);
  random_matrix_double(b, matrixDim, matrixDim);
  #else
  random_matrix_float(a, matrixDim, matrixDim);
  random_matrix_float(b, matrixDim, matrixDim);
  #endif

  // copy inputs to device
  HANDLE_ERROR(hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice));

  // grid settings
  gridSize = matrixDim / blockSize;
  if (gridSize * blockSize < matrixDim) {
     gridSize += 1;
  }
  blockDim.x = blockSize;
  blockDim.y = blockSize;
  gridDim.x = gridSize;
  gridDim.y = gridSize;

  // launch add() kernel
  add<<< gridDim, blockDim >>>(dev_a, dev_b, dev_c, matrixDim);

  // copy device result back to host copy of c
  HANDLE_ERROR(hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost));

  // test result
  REAL *expected;
  HANDLE_NULL(expected = (REAL*)malloc(size));
  #ifdef DOUBLE
  matrix_add_double(a, b, expected, matrixDim, matrixDim);
  const bool equal = matrix_equals_double(c, expected, matrixDim, matrixDim);
  #else
  matrix_add_float(a, b, expected, matrixDim, matrixDim);
  const bool equal = matrix_equals_float(c, expected, matrixDim, matrixDim);
  #endif
  if (!equal) {
    fprintf(stderr, "Error\n");
  } else {
    printf("Correct\n");
  }

  // free host
  free(a);
  free(b);
  free(c);
  free(expected);

  // free device
  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipFree(dev_b));
  HANDLE_ERROR(hipFree(dev_c));

  return 0;
}
