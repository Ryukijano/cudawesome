#include "hip/hip_runtime.h"
/*
 * @Name: matrix_add_nxn_int.cu
 * @Description: Matrix (NxN) Integer Sum
 * Each matrix is viewed as a single block of memory.
 * Blocks and threads are viewed as a 2D grid.
 * Custom matrix dimension and block size.
 *
 * @Author: Giacomo Marciani <gmarciani@acm.org>
 * @Institution: University of Rome Tor Vergata
 *
 * @Usage: matrix_add_nxn_int matrixDim blockSize
 *
 * Default values:
 *  matrixDim: 4096
 *  blockSize: 32
 */

#include <stdio.h>
#include <math.h>
#include "../../common/error.h"
#include "../../common/random.h"
#include "../../common/matrix.h"

__global__ void matrixAdd(const int *a, const int *b, int *c, const unsigned int dim) {
  const unsigned int iX = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int iY = blockIdx.y * blockDim.y + threadIdx.y;

  if (iX >= dim || iY >= dim) return;

  const unsigned int pos = iY * dim + iX;
  c[pos] = a[pos] + b[pos];
}

__host__ void gpuMatrixAdd(const int *a, const int *b, int *c, const unsigned int matrixDim, const dim3 gridDim, const dim3 blockDim) {
  int *dev_a, *dev_b, *dev_c; // device copies of a, b, c
  const unsigned int size = matrixDim * matrixDim * sizeof(int); // bytes for a, b, c

  // allocate device copies of a, b, c
  HANDLE_ERROR(hipMalloc((void**)&dev_a, size));
  HANDLE_ERROR(hipMalloc((void**)&dev_b, size));
  HANDLE_ERROR(hipMalloc((void**)&dev_c, size));

  // copy inputs to device
  HANDLE_ERROR(hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice));

  // launch kernel matrixAdd()
  matrixAdd<<< gridDim, blockDim >>>(dev_a, dev_b, dev_c, matrixDim);

  // copy device result back to host copy of c
  HANDLE_ERROR(hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost));

  // free device
  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipFree(dev_b));
  HANDLE_ERROR(hipFree(dev_c));
}

int main(const int argc, const char **argv) {
  int *a, *b, *c;    // host copies of a, b, c
  unsigned int size; // bytes for a, b, c
  unsigned int matrixDim; // matrix dimensions
  unsigned int gridSize; // grid size
  unsigned int blockSize; // block size
  hipDeviceProp_t gpuInfo; // gpu properties

  // check arguments
  if (argc < 3) {
    fprintf(stderr, "Usage: %s matrixDim blockSize\n", argv[0]);
    exit(1);
  }

  matrixDim = atoi(argv[1]);
  blockSize = atoi(argv[2]);

  if (matrixDim < 1) {
    fprintf(stderr, "Error: matrixDim expected >= 1, got %d\n", matrixDim);
    exit(1);
  }

  if (blockSize < 1) {
    fprintf(stderr, "Error: blockSize expected >= 1, got %d\n", blockSize);
    exit(1);
  }

  // grid settings
  gridSize = matrixDim / blockSize;
  if (gridSize * blockSize < matrixDim) {
     gridSize += 1;
  }
  dim3 gridDim(gridSize, gridSize);
  dim3 blockDim(blockSize, blockSize);

  size = matrixDim * matrixDim * sizeof(int);

  HANDLE_ERROR(hipGetDeviceProperties(&gpuInfo, 0));

  printf("------------------------------------\n");
  printf("Matrix (NxN) Integer Sum\n");
  printf("------------------------------------\n");
  printf("Matrix Dimension: (%d, %d)\n", matrixDim, matrixDim);
  printf("Grid Size: (%d, %d, %d) (max: (%d, %d, %d))\n",
    gridDim.x, gridDim.y, gridDim.z,
    gpuInfo.maxGridSize[0], gpuInfo.maxGridSize[1], gpuInfo.maxGridSize[2]);
  printf("Block Size: (%d, %d, %d) (max: (%d, %d, %d))\n",
    blockDim.x, blockDim.y, blockDim.z,
    gpuInfo.maxThreadsDim[0], gpuInfo.maxThreadsDim[1], gpuInfo.maxThreadsDim[2]);
  printf("-----------------------------------\n");

  // allocate host copies of a, b, c
  HANDLE_NULL(a = (int*)malloc(size));
  HANDLE_NULL(b = (int*)malloc(size));
  HANDLE_NULL(c = (int*)malloc(size));

  // fill a, b with random data
  random_matrix_int(a, matrixDim, matrixDim);
  random_matrix_int(b, matrixDim, matrixDim);

  // launch kernel matrixAdd()
  gpuMatrixAdd(a, b, c, matrixDim, gridDim, blockDim);

  // test result
  int *expected;
  HANDLE_NULL(expected = (int*)malloc(size));
  matrix_add_int(a, b, expected, matrixDim, matrixDim);
  const bool equal = matrix_equals_int(c, expected, matrixDim, matrixDim);
  if (!equal) {
    fprintf(stderr, "Error\n");
  } else {
    printf("Correct\n");
  }

  // free host
  free(a);
  free(b);
  free(c);
  free(expected);

  return 0;
}
