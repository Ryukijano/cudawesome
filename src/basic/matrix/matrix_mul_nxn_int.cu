#include "hip/hip_runtime.h"
/*
 * @Name: matrix_mul_nxn_int.cu
 * @Description: Matrix (NxN) Integer Product.
 * Each matrix is viewed as a single block of memory.
 * Blocks and threads are viewed as a 2D grid.
 * Custom matrix dimension and block size.
 *
 * @Author: Giacomo Marciani <gmarciani@acm.org>
 * @Institution: University of Rome Tor Vergata
 *
 * @Usage: matrix_mul_nxn_int matrixDim blockSize
 *
 * Default values:
 *  matrixDim: 4096
 *  blockSize: 32
 */

#include <stdio.h>
#include <math.h>
#include "../../common/error.h"
#include "../../common/random.h"
#include "../../common/matrix.h"
#include "../../common/mathutil.h"

__global__ void matrixMul(const int *a, const int *b, int *c, const unsigned int dim) {
  const unsigned int iX = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int iY = blockIdx.y * blockDim.y + threadIdx.y;

  if (iX >= dim || iY >= dim) return;

  const unsigned int pos = iY * dim + iX;

  int val = 0;
  for (unsigned int k = 0; k < dim; k++) {
    val += a[iY * dim + k] * b[k * dim + iX];
  }

  c[pos] = val;
}

__host__ void gpuMatrixMul(const int *a, const int *b, int *c, const unsigned int matrixDim, const dim3 gridDim, const dim3 blockDim) {
  int *dev_a, *dev_b, *dev_c; // device copies of a, b, c
  const unsigned int size = matrixDim * matrixDim * sizeof(int); // bytes for a, b, c

  // allocate device copy of a, b, c
  HANDLE_ERROR(hipMalloc((void**)&dev_a, size));
  HANDLE_ERROR(hipMalloc((void**)&dev_b, size));
  HANDLE_ERROR(hipMalloc((void**)&dev_c, size));

  // copy inputs to device
  HANDLE_ERROR(hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice));

  // launch mul() kernel
  matrixMul<<< gridDim, blockDim >>>(dev_a, dev_b, dev_c, matrixDim);

  // copy device result back to host copy of c
  HANDLE_ERROR(hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost));

  // free device
  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipFree(dev_b));
  HANDLE_ERROR(hipFree(dev_c));
}

int main(const int argc, const char **argv) {
  int *a, *b, *c;         // host copies of a, b, c
  unsigned int size; // bytes for a, b, c
  unsigned int matrixDim; // matrices dimensions
  unsigned int gridSizeX, gridSizeY; // grid size
  unsigned int blockSize; // block size
  hipDeviceProp_t gpuInfo; // gpu properties

  // check arguments
  if (argc < 3) {
    fprintf(stderr, "Usage: %s matrixDim blockSize\n", argv[0]);
    exit(1);
  }

  matrixDim = atoi(argv[1]);
  blockSize = atoi(argv[2]);

  if (matrixDim < 1) {
    fprintf(stderr, "Error: matrixDim expected >= 1, got %d\n", matrixDim);
    exit(1);
  }

  if (!IS_POWER_OF_2(blockSize)) {
    fprintf(stderr, "Error: blockSize expected as power of 2, got %d\n", blockSize);
    exit(1);
  }

  // grid settings
  gridSizeX = matrixDim / blockSize;
  if (gridSizeX * blockSize < matrixDim) {
     gridSizeX += 1;
  }
  gridSizeY = matrixDim / blockSize;
  if (gridSizeY * blockSize < matrixDim) {
     gridSizeY += 1;
  }
  dim3 gridDim(gridSizeX, gridSizeY);
  dim3 blockDim(blockSize, blockSize);

  size = matrixDim * matrixDim * sizeof(int);

  HANDLE_ERROR(hipGetDeviceProperties(&gpuInfo, 0));

  printf("------------------------------------\n");
  printf("Matrix (NxM) Integer Product\n");
  printf("------------------------------------\n");
  printf("Matrix Dimension (A): (%d, %d)\n", matrixDim, matrixDim);
  printf("Matrix Dimension (B): (%d, %d)\n", matrixDim, matrixDim);
  printf("Grid Size: (%d, %d, %d) (max: (%d, %d, %d))\n",
    gridDim.x, gridDim.y, gridDim.z,
    gpuInfo.maxGridSize[0], gpuInfo.maxGridSize[1], gpuInfo.maxGridSize[2]);
  printf("Block Size: (%d, %d, %d) (max: (%d, %d, %d))\n",
    blockDim.x, blockDim.y, blockDim.z,
    gpuInfo.maxThreadsDim[0], gpuInfo.maxThreadsDim[1], gpuInfo.maxThreadsDim[2]);
  printf("-----------------------------------\n");

  // allocate host copy of a, b, c
  HANDLE_NULL(a = (int*)malloc(size));
  HANDLE_NULL(b = (int*)malloc(size));
  HANDLE_NULL(c = (int*)malloc(size));

  // fill a, b with random data
  random_matrix_int(a, matrixDim, matrixDim);
  random_matrix_int(b, matrixDim, matrixDim);

  // launch kernel matrixMul()
  gpuMatrixMul(a, b, c, matrixDim, gridDim, blockDim);

  // test result
  int *expected;
  HANDLE_NULL(expected = (int*)malloc(size));
  matrix_mul_int(a, b, expected, matrixDim, matrixDim, matrixDim);
  const bool correct = matrix_equals_int(c, expected, matrixDim, matrixDim);
  if (!correct) {
    fprintf(stderr, "Error\n");
  } else {
    printf("Correct\n");
  }

  // free host
  free(a);
  free(b);
  free(c);
  free(expected);

  return 0;
}
