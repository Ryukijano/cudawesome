#include "hip/hip_runtime.h"
/*
 * @Name: matrix_mul_nxn_int.cu
 * @Description: Matrix (NxN) Integer Product.
 * Each matrix is viewed as a single block of memory.
 * Blocks and threads are viewed as a 2D grid.
 * Custom matrix dimension and block size.
 *
 * @Author: Giacomo Marciani <gmarciani@acm.org>
 * @Institution: University of Rome Tor Vergata
 *
 * @Usage: matrix_mul_nxn_int matrixDim blockSize
 *
 * Default values:
 *  matrixDim: 4096
 *  blockSize: 32
 */

#include <stdio.h>
#include <math.h>
#include "../../common/error.h"
#include "../../common/random.h"
#include "../../common/matrix.h"
#include "../../common/mathutil.h"

__global__ void mul(const int *a, const int *b, int *c, const unsigned int dim) {
  const unsigned int iX = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int iY = blockIdx.y * blockDim.y + threadIdx.y;

  if (iX < dim && iY < dim) {
    const unsigned int pos = iY * dim + iX;
    int val = 0;
    for (unsigned int k = 0; k < dim; k++) {
      val += a[iY * dim + k] * b[k * dim + iX];
    }

    c[pos] = val;
  }
}

int main(const int argc, const char **argv) {
  int *a, *b, *c;         // host copies of a, b, c
  int *dev_a, *dev_b, *dev_c; // device copies of a, b, c
  unsigned int size; // bytes for a, b, c
  unsigned int matrixDim; // matrix dimension
  dim3 gridDim;  // grid dimension
  dim3 blockDim; // block dimension
  unsigned int gridSize; // grid size
  unsigned int blockSize; // block size
  hipDeviceProp_t gpuInfo; // gpu properties

  if (argc < 3) {
    fprintf(stderr, "Usage: %s matrixDim blockSize\n", argv[0]);
    exit(1);
  }

  matrixDim = atoi(argv[1]);
  blockSize = atoi(argv[2]);

  if (matrixDim < 1) {
    fprintf(stderr, "Error: matrixDim expected >= 1, got %d\n", matrixDim);
    exit(1);
  }

  if (!IS_POWER_OF_2(blockSize)) {
    fprintf(stderr, "Error: blockSize expected as power of 2, got %d\n", blockSize);
    exit(1);
  }

  // grid settings
  gridSize = matrixDim / blockSize;
  if (gridSize * blockSize < matrixDim) {
     gridSize += 1;
  }
  blockDim.x = blockSize;
  blockDim.y = blockSize;
  gridDim.x = gridSize;
  gridDim.y = gridSize;

  size = matrixDim * matrixDim * sizeof(int);

  HANDLE_ERROR(hipGetDeviceProperties(&gpuInfo, 0));

  printf("------------------------------------\n");
  printf("Matrix (NxN) Integer Product\n");
  printf("------------------------------------\n");
  printf("Matrix Dimension (A): (%d, %d)\n", matrixDim, matrixDim);
  printf("Matrix Dimension (B): (%d, %d)\n", matrixDim, matrixDim);
  printf("Grid Size: (%d, %d, %d) (max: (%d, %d, %d))\n",
    gridDim.x, gridDim.y, gridDim.z,
    gpuInfo.maxGridSize[0], gpuInfo.maxGridSize[1], gpuInfo.maxGridSize[2]);
  printf("Block Size: (%d, %d, %d) (max: (%d, %d, %d))\n",
    blockDim.x, blockDim.y, blockDim.z,
    gpuInfo.maxThreadsDim[0], gpuInfo.maxThreadsDim[1], gpuInfo.maxThreadsDim[2]);
  printf("-----------------------------------\n");

  // allocate host copy of a, b, c
  HANDLE_NULL(a = (int*)malloc(size));
  HANDLE_NULL(b = (int*)malloc(size));
  HANDLE_NULL(c = (int*)malloc(size));

  // allocate device copy of a, b, c
  HANDLE_ERROR(hipMalloc((void**)&dev_a, size));
  HANDLE_ERROR(hipMalloc((void**)&dev_b, size));
  HANDLE_ERROR(hipMalloc((void**)&dev_c, size));

  // fill a, b with random data
  random_matrix_int(a, matrixDim, matrixDim);
  random_matrix_int(b, matrixDim, matrixDim);

  // copy inputs to device
  HANDLE_ERROR(hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice));

  // launch mul() kernel
  mul<<< gridDim, blockDim >>>(dev_a, dev_b, dev_c, matrixDim);

  // copy device result back to host copy of c
  HANDLE_ERROR(hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost));

  // test result
  int *expected;
  HANDLE_NULL(expected = (int*)malloc(size));
  matrix_mul_int(a, b, expected, matrixDim, matrixDim, matrixDim);
  if (!matrix_equals_int(c, expected, matrixDim, matrixDim)) {
    fprintf(stderr, "Error\n");
  } else {
    printf("Correct\n");
  }

  // free host
  free(a);
  free(b);
  free(c);
  free(expected);

  // free device
  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipFree(dev_b));
  HANDLE_ERROR(hipFree(dev_c));

  return 0;
}
