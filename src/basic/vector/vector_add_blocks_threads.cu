#include "hip/hip_runtime.h"
/*
 * @Name: vector_add_blocks_threads.cu
 * @Description: Addition of two integer vectors.
 * Multiple blocks, multiple threads per block.
 *
 * @Author: Giacomo Marciani <gmarciani@acm.org>
 * @Institution: University of Rome Tor Vergata
 */

#include <stdio.h>
#include <math.h>
#include <../common/error.h>
#include <../common/random.h>
#include <../common/vector.h>

#define VECTOR_DIM 512
#define BLOCK_SIZE 16

__global__ void add(int *a, int *b, int *c) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  c[idx] = a[idx] + b[idx];
}

int main(void) {
  int *a, *b, *c;             // host copies of a, b, c
  int *dev_a, *dev_b, *dev_c; // device copies of a, b, c
  int size = VECTOR_DIM * sizeof(int); // bytes for an array of VECTOR_DIM integers

  // allocate host copies of a, b, c
  a = HANDLE_NULL((int*)malloc(size));
  b = HANDLE_NULL((int*)malloc(size));
  c = HANDLE_NULL((int*)malloc(size));

  // allocate device copies of a, b, c
  HANDLE_ERROR(hipMalloc((void**)&dev_a, size));
  HANDLE_ERROR(hipMalloc((void**)&dev_b, size));
  HANDLE_ERROR(hipMalloc((void**)&dev_c, size));

  // fill a and b with VECTOR_DIM random integers
  random_ints(a, VECTOR_DIM);
  random_ints(b, VECTOR_DIM);

  // copy inputs to device
  HANDLE_ERROR(hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice));

  // launch add() kernel
  add<<< VECTOR_DIM / BLOCK_SIZE, BLOCK_SIZE >>>(dev_a, dev_b, dev_c);

  // copy device result back to host copy of c
  HANDLE_ERROR(hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost));

  // test result
  int *d = HANDLE_NULL((int*)malloc(size));
  vector_add(a, b, d, vectorDim);
  for (int i = 0; i < vectorDim; i++) {
    if (c[i] != d[i]) {
      printf("Error: [%d] expected %d, got %d\n", i, d, c[i]);
      break;
    }
  }

  // free host
  free(a);
  free(b);
  free(c);
  free(d);

  // free device
  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipFree(dev_b));
  HANDLE_ERROR(hipFree(dev_c));

  return 0;
}
