#include "hip/hip_runtime.h"
/*
 * @Name: vector_add_threads.cu
 * @Description: Addition of two integer vectors.
 * One block, multiple threads per block.
 *
 * @Author: Giacomo Marciani <gmarciani@acm.org>
 * @Institution: University of Rome Tor Vergata
 */

#include <stdio.h>
#include <math.h>
#include "../../common/error.h"
#include "../../common/random.h"
#include "../../common/vector.h"

#define VECTOR_DIM 512

__global__ void add(double *a, double *b, double *c) {
  c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

int main(void) {
  double *a, *b, *c;             // host copies of a, b, c
  double *dev_a, *dev_b, *dev_c; // device copies of a, b, c
  int size = VECTOR_DIM * sizeof(double); // bytes for a, b, c

  // allocate host copies of a, b, c
  HANDLE_NULL(a = (double*)malloc(size));
  HANDLE_NULL(b = (double*)malloc(size));
  HANDLE_NULL(c = (double*)malloc(size));

  // allocate device copies of a, b, c
  HANDLE_ERROR(hipMalloc((void**)&dev_a, size));
  HANDLE_ERROR(hipMalloc((void**)&dev_b, size));
  HANDLE_ERROR(hipMalloc((void**)&dev_c, size));

  // fill a, b with random data
  random_vector_double(a, VECTOR_DIM);
  random_vector_double(b, VECTOR_DIM);

  // copy inputs to device
  HANDLE_ERROR(hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice));

  // launch add() kernel
  add<<< 1, VECTOR_DIM >>>(dev_a, dev_b, dev_c);

  // copy device result back to host copy of c
  HANDLE_ERROR(hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost));

  // test result
  double *d;
  HANDLE_NULL(d = (double*)malloc(size));
  vector_add_double(a, b, d, VECTOR_DIM);
  if (!vector_equals_double(c, d, VECTOR_DIM)) {
    fprintf(stderr, "Error\n");
  } else {
    printf("Correct\n");
  }

  // free host
  free(a);
  free(b);
  free(c);
  free(d);

  // free device
  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipFree(dev_b));
  HANDLE_ERROR(hipFree(dev_c));

  return 0;
}
