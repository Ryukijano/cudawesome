#include "hip/hip_runtime.h"
/*
 * @Name: vector_dot_int_3.cu
 * @Description: Integer vectors dot-product.
 * Multiple blocks, multiple threads per block.
 *
 * @Author: Giacomo Marciani <gmarciani@acm.org>
 * @Institution: University of Rome Tor Vergata
 *
 * @Usage: vector_dot_int_3 vectorDimension blockSize
 *
 * Default values:
 *  vectorDimension: 4096
 *  blockSize: 32
 *
 * WARNING: works only if (vectorDim % blockSize) == 0
 *
 * @See: http://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/reduction/doc/reduction.pdf
 */

#include <stdio.h>
#include <math.h>
#include "../../common/error.h"
#include "../../common/random.h"
#include "../../common/vector.h"
#include "../../common/mathutil.h"

__global__ void dot(const int *a, const int *b, int *c, const unsigned int vectorDim) {
  extern __shared__ int temp[];

  const unsigned int tid = threadIdx.x;
  const unsigned int pos = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

  if (pos + blockDim.x < vectorDim) {
    temp[tid] = (a[pos] * b[pos]) + (a[pos + blockDim.x] * b[pos + blockDim.x]);

    __syncthreads();

    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
      if (tid < stride) {
        temp[tid] += temp[tid + stride];
      }
      __syncthreads();
    }

    if (0 == tid) {
      c[blockIdx.x] = temp[0];
    }
  }
}

int main(const int argc, const char **argv) {
  int *a, *b, *c, result;     // host copies of a, b, c, result
  int *dev_a, *dev_b, *dev_c; // device copies of a, b, c
  unsigned int size_a_b; // bytes for a, b
  unsigned int size_c; // bytes for c
  unsigned int vectorDim; // vector dimension
  unsigned int gridSize;  // grid size
  unsigned int blockSize; // block size
  hipDeviceProp_t gpuInfo; // gpu properties

  // check arguments
  if (argc < 3) {
    fprintf(stderr, "Usage: %s vectorDim blockSize\n", argv[0]);
    exit(1);
  }

  vectorDim = atoi(argv[1]);
  blockSize = atoi(argv[2]);

  if (vectorDim < 1) {
    fprintf(stderr, "Error: vectorDim expected >= 1, got %d\n", vectorDim);
    exit(1);
  }

  if (!IS_POWER_OF_2(blockSize)) {
    fprintf(stderr, "Error: blockSize expected as power of 2, got %d\n", blockSize);
    exit(1);
  }

  HANDLE_ERROR(hipGetDeviceProperties(&gpuInfo, 0));

  gridSize = vectorDim / blockSize;
  if (gridSize * blockSize < vectorDim) {
    gridSize += 1;
  }

  size_a_b = vectorDim * sizeof(int);
  size_c = gridSize * sizeof(int);

  printf("----------------------------------------------\n");
  printf("Vector Integer Dot Product\n");
  printf("Reduction: sequential addressing (add-on-load)\n");
  printf("----------------------------------------------\n");
  printf("Vector Dimension: %d\n", vectorDim);
  printf("Grid Size: %d (max: %d)\n", gridSize, gpuInfo.maxGridSize[0]);
  printf("Block Size: %d (max: %d)\n", blockSize, gpuInfo.maxThreadsDim[1]);
  printf("----------------------------------------------\n");

  // allocate host copies of a, b, c
  HANDLE_NULL(a = (int*)malloc(size_a_b));
  HANDLE_NULL(b = (int*)malloc(size_a_b));
  HANDLE_NULL(c = (int*)malloc(size_c));

  // allocate device copies of a, b, c
  HANDLE_ERROR(hipMalloc((void**)&dev_a, size_a_b));
  HANDLE_ERROR(hipMalloc((void**)&dev_b, size_a_b));
  HANDLE_ERROR(hipMalloc((void**)&dev_c, size_c));

  // fill a, b with random data
  random_vector_int(a, vectorDim);
  random_vector_int(b, vectorDim);

  // copy inputs to device
  HANDLE_ERROR(hipMemcpy(dev_a, a, size_a_b, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, size_a_b, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemset(dev_c, 0, size_c));

  // shared memory settings
  const unsigned int sharedMemSize = (unsigned int) gridSize * sizeof(int);

  // launch dot() kernel
  dot<<< gridSize, blockSize, sharedMemSize >>>(dev_a, dev_b, dev_c, vectorDim);

  // copy device result back to host copy of c
  HANDLE_ERROR(hipMemcpy(c, dev_c, size_c, hipMemcpyDeviceToHost));

  // reduce blocks result
  result = 0;
  for (unsigned int block = 0; block < gridSize; block++) {
    result += c[block];
  }

  // test result
  int expected;
  vector_dot_int(a, b, &expected, vectorDim);
  if (result != expected) {
    fprintf(stderr, "Error: expected %d, got %d (error:%f %%)\n",
      expected, result, (((float)result - (float)expected) / (float)expected) * 100.0);
  } else {
    printf("Correct\n");
  }

  // free host
  free(a);
  free(b);
  free(c);

  // free device
  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipFree(dev_b));
  HANDLE_ERROR(hipFree(dev_c));

  return 0;
}
