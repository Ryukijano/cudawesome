#include "hip/hip_runtime.h"
/*
 * @Name: vector_dot_int_3.cu
 * @Description: Vector Integer Dot Product.
 * Multiple blocks, multiple threads per block.
 *
 * @Author: Giacomo Marciani <gmarciani@acm.org>
 * @Institution: University of Rome Tor Vergata
 *
 * @Usage: vector_dot_int_3 vectorDimension blockSize
 *
 * Default values:
 *  vectorDimension: 4096
 *  blockSize: 32
 *
 * WARNING: works only if (vectorDim % blockSize) == 0
 *
 * @See: http://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/reduction/doc/reduction.pdf
 */

#include <stdio.h>
#include <math.h>
#include "../../common/error.h"
#include "../../common/random.h"
#include "../../common/vector.h"
#include "../../common/mathutil.h"

__global__ void vectorDot(const int *a, const int *b, int *c, const unsigned int vectorDim) {
  extern __shared__ int temp[];

  const unsigned int tid = threadIdx.x;
  const unsigned int pos = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

  if (pos + blockDim.x >= vectorDim) return;

  temp[tid] = (a[pos] * b[pos]) + (a[pos + blockDim.x] * b[pos + blockDim.x]);

  __syncthreads();

  for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
    if (tid < stride) {
      temp[tid] += temp[tid + stride];
    }
    __syncthreads();
  }

  if (0 == tid) {
    c[blockIdx.x] = temp[0];
  }
}

__host__ void gpuVectorDot(const int *a, const int *b, int *result, const unsigned int vectorDim, const dim3 gridDim, const dim3 blockDim) {
  int *dev_a, *dev_b, *dev_partial; // device copies of a, b, partial
  int *partial; // host copy for partial result
  const unsigned int size_a_b = vectorDim * sizeof(int); // bytes for a, b
  const unsigned int size_partial = gridDim.x * sizeof(int); // bytes for partial

  // allocate host copies of partial
  HANDLE_NULL(partial = (int*)malloc(size_partial));

  // allocate device copies of a, b, c
  HANDLE_ERROR(hipMalloc((void**)&dev_a, size_a_b));
  HANDLE_ERROR(hipMalloc((void**)&dev_b, size_a_b));
  HANDLE_ERROR(hipMalloc((void**)&dev_partial, size_partial));

  // copy inputs to device
  HANDLE_ERROR(hipMemcpy(dev_a, a, size_a_b, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, size_a_b, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemset(dev_partial, 0, size_partial));

  // shared memory settings
  const unsigned int sharedMemSize = (unsigned int) blockDim.x * sizeof(int);

  // launch kernel vectorDot
  vectorDot<<< gridDim, blockDim, sharedMemSize >>>(dev_a, dev_b, dev_partial, vectorDim);

  // copy device result back to host copy of c
  HANDLE_ERROR(hipMemcpy(partial, dev_partial, size_partial, hipMemcpyDeviceToHost));

  // reduce blocks result
  *result = 0;
  for (unsigned int block = 0; block < gridDim.x; block++) {
    (*result) += partial[block];
  }

  // free host
  free(partial);

  // free device
  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipFree(dev_b));
  HANDLE_ERROR(hipFree(dev_partial));
}

int main(const int argc, const char **argv) {
  int *a, *b, result; // host copies of a, b, result
  unsigned int vectorDim; // vector dimension
  unsigned int size_a_b; // bytes for a, b
  unsigned int gridSize;  // grid size
  unsigned int blockSize; // block size
  hipDeviceProp_t gpuInfo; // gpu properties

  // check arguments
  if (argc < 3) {
    fprintf(stderr, "Usage: %s vectorDim blockSize\n", argv[0]);
    exit(1);
  }

  vectorDim = atoi(argv[1]);
  blockSize = atoi(argv[2]);

  if (vectorDim < 1) {
    fprintf(stderr, "Error: vectorDim expected >= 1, got %d\n", vectorDim);
    exit(1);
  }

  if (!IS_POWER_OF_2(blockSize)) {
    fprintf(stderr, "Error: blockSize expected as power of 2, got %d\n", blockSize);
    exit(1);
  }

  // grid settings
  gridSize = vectorDim / blockSize;
  if (gridSize * blockSize < vectorDim) {
    gridSize += 1;
  }
  dim3 gridDim(gridSize);
  dim3 blockDim(blockSize);

  size_a_b = vectorDim * sizeof(int);

  HANDLE_ERROR(hipGetDeviceProperties(&gpuInfo, 0));

  printf("----------------------------------\n");
  printf("Vector Integer Dot Product\n");
  printf("Reduction: sequential addressing (add-on-load)\n");
  printf("----------------------------------\n");
  printf("Vector Dimension: %d\n", vectorDim);
  printf("Grid Size: (%d %d %d) (max: (%d %d %d))\n",
    gridDim.x, gridDim.y, gridDim.z,
    gpuInfo.maxGridSize[0], gpuInfo.maxGridSize[1], gpuInfo.maxGridSize[2]);
  printf("Block Size: (%d %d %d) (max: (%d %d %d))\n",
    blockDim.x, blockDim.y, blockDim.z,
    gpuInfo.maxThreadsDim[0], gpuInfo.maxThreadsDim[1], gpuInfo.maxThreadsDim[2]);
  printf("---------------------------------\n");

  // allocate host copies of a, b, c
  HANDLE_NULL(a = (int*)malloc(size_a_b));
  HANDLE_NULL(b = (int*)malloc(size_a_b));

  // fill a, b with random data
  random_vector_int(a, vectorDim);
  random_vector_int(b, vectorDim);

  // launch kernel vectorDot()
  gpuVectorDot(a, b, &result, vectorDim, gridDim, blockDim);

  // test result
  int expected;
  vector_dot_int(a, b, &expected, vectorDim);
  if (result != expected) {
    fprintf(stderr, "Error: expected %d, got %d (error:%f %%)\n",
      expected, result, (abs((float)expected - (float)result) / (float)expected) * 100.0);
  } else {
    printf("Correct\n");
  }

  // free host
  free(a);
  free(b);

  return 0;
}
