#include "hip/hip_runtime.h"
/*
 * @Name: vector_add_int.cu
 * @Description: Vector Integer Sum.
 * Custom vector dimension and block size.
 *
 * @Author: Giacomo Marciani <gmarciani@acm.org>
 * @Institution: University of Rome Tor Vergata
 *
 * @Usage: vector_add_int vectorDimension blockSize
 *
 * Default values:
 *  vectorDimension: 4096
 *  blockSize: 32
 */

#include <stdio.h>
#include <math.h>
#include "../../common/error.h"
#include "../../common/random.h"
#include "../../common/vector.h"
#include "../../common/mathutil.h"

__global__ void vectorAdd(const int *a, const int *b, int *c, const unsigned int dim) {
  const unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;

  if (pos >= dim) return;

  c[pos] = a[pos] + b[pos];
}

__host__ void gpuVectorAdd(const int *a, const int *b, int *c, const unsigned int vectorDim, const dim3 gridDim, const dim3 blockDim) {
  int *dev_a, *dev_b, *dev_c; // device copies of a, b, c
  const unsigned int size = vectorDim * sizeof(int); // bytes for a, b, c

  // allocate device copies of a, b, c
  HANDLE_ERROR(hipMalloc((void**)&dev_a, size));
  HANDLE_ERROR(hipMalloc((void**)&dev_b, size));
  HANDLE_ERROR(hipMalloc((void**)&dev_c, size));

  // copy inputs to device
  HANDLE_ERROR(hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice));

  // launch kernel vectorAdd()
  vectorAdd<<< gridDim, blockDim >>>(dev_a, dev_b, dev_c, vectorDim);

  // copy device result back to host copy of c
  HANDLE_ERROR(hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost));

  // free device
  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipFree(dev_b));
  HANDLE_ERROR(hipFree(dev_c));
}

int main(const int argc, const char **argv) {
  int *a, *b, *c;    // host copies of a, b, c
  unsigned int size; // bytes for a, b, c
  unsigned int vectorDim; // vector dimension
  unsigned int gridSize;  // grid size
  unsigned int blockSize; // block size
  hipDeviceProp_t gpuInfo; // gpu properties

  // check arguments
  if (argc < 3) {
    fprintf(stderr, "Usage: %s vectorDim blockSize\n", argv[0]);
    exit(1);
  }

  vectorDim = atoi(argv[1]);
  blockSize = atoi(argv[2]);

  if (vectorDim < 1) {
    fprintf(stderr, "Error: vectorDim expected >= 1, got %d\n", vectorDim);
    exit(1);
  }

  if (!IS_POWER_OF_2(blockSize)) {
    fprintf(stderr, "Error: blockSize expected as power of 2, got %d\n", blockSize);
    exit(1);
  }

  // grid settings
  gridSize = vectorDim / blockSize;
  if (gridSize * blockSize < vectorDim) {
    gridSize += 1;
  }
  dim3 gridDim(gridSize);
  dim3 blockDim(blockSize);

  size = vectorDim * sizeof(int);

  HANDLE_ERROR(hipGetDeviceProperties(&gpuInfo, 0));

  printf("----------------------------------\n");
  printf("Vector Integer Sum\n");
  printf("----------------------------------\n");
  printf("Vector Dimension: %d\n", vectorDim);
  printf("Grid Size: (%d %d %d) (max: (%d %d %d))\n",
    gridDim.x, gridDim.y, gridDim.z,
    gpuInfo.maxGridSize[0], gpuInfo.maxGridSize[1], gpuInfo.maxGridSize[2]);
  printf("Block Size: (%d %d %d) (max: (%d %d %d))\n",
    blockDim.x, blockDim.y, blockDim.z,
    gpuInfo.maxThreadsDim[0], gpuInfo.maxThreadsDim[1], gpuInfo.maxThreadsDim[2]);
  printf("---------------------------------\n");

  // allocate host copies of a, b, c
  HANDLE_NULL(a = (int*)malloc(size));
  HANDLE_NULL(b = (int*)malloc(size));
  HANDLE_NULL(c = (int*)malloc(size));

  // fill a, b with random data
  random_vector_int(a, vectorDim);
  random_vector_int(b, vectorDim);

  // launch kernel vectorAdd()
  gpuVectorAdd(a, b, c, vectorDim, gridDim, blockDim);

  // test result
  int *expected;
  HANDLE_NULL(expected = (int*)malloc(size));
  vector_add_int(a, b, expected, vectorDim);
  const bool correct = vector_equals_int(c, expected, vectorDim);
  if (!correct) {
    fprintf(stderr, "Error\n");
  } else {
    printf("Correct\n");
  }

  // free host
  free(a);
  free(b);
  free(c);
  free(expected);

  return 0;
}
