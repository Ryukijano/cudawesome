#include "hip/hip_runtime.h"
/*
 * @Name: vector_dot_threads.cu
 * @Description: Integer vectors dot-product.
 * One block, multiple threads per block.
 *
 * @Author: Giacomo Marciani <gmarciani@acm.org>
 * @Institution: University of Rome Tor Vergata
 */

#include <stdio.h>
#include <math.h>
#include <../common/error.h>
#include <../common/random.h>

#define VECTOR_DIM 512

__global__ void dot(int *a, int *b, int *c) {
  __shared__ int temp[N];

  temp[threadIdx.x] = a[threadIdx.x] * b[threadIdx.x];

  __syncthreads();

  if (0 == threadIdx.x) {
    int sum = 0;
    for (int i = N - 1; i >=0; i--) {
      sum += temp[i];
    }
  }

  *c = sum;
}

int main(void) {
  int *a, *b, c;              // host copies of a, b, c
  int *dev_a, *dev_b, *dev_c; // device copies of a, b, c
  int size = VECTOR_DIM * sizeof(int); // bytes for an array of VECTOR_DIM integers

  // allocate host copies of a, b, c
  a = HANDLE_NULL((int*)malloc(size));
  b = HANDLE_NULL((int*)malloc(size));
  c = HANDLE_NULL((int*)malloc(sizeof(int)));

  // allocate device copies of a, b, c
  HANDLE_ERROR(hipMalloc((void**)&dev_a, size));
  HANDLE_ERROR(hipMalloc((void**)&dev_b, size));
  HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int)));

  // fill a and b with VECTOR_DIM random integers
  random_ints(a, VECTOR_DIM);
  random_ints(b, VECTOR_DIM);

  // copy inputs to device
  HANDLE_ERROR(hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice));

  // launch add() kernel
  dot<<< 1, VECTOR_DIM >>>(dev_a, dev_b, dev_c);

  // copy device result back to host copy of c
  HANDLE_ERROR(hipMemcpy(c, dev_c, sizeof(int), hipMemcpyDeviceToHost));

  // test result
  int d = 0;
  for(int i = 0; i < N; i++) {
    d += a[i] * b[i];
  }
  if (*c != d) {
    printf("Error: expected %d, got %d\n", d, *c);
  }

  // free host
  free(a);
  free(b);
  free(c);

  // free device
  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipFree(dev_b));
  HANDLE_ERROR(hipFree(dev_c));

  return 0;
}
