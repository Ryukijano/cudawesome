#include "hip/hip_runtime.h"
/*
 * @Name: vector_add.cu
 * @Description: Addition of two integer vectors.
 * Custom vector dimension and block size.
 *
 * @Author: Giacomo Marciani <gmarciani@acm.org>
 * @Institution: University of Rome Tor Vergata
 *
 * @Usage: vector_add vectorDimension blockSize
 */

#include <stdio.h>
#include <math.h>
#include "../../common/error.h"
#include "../../common/random.h"
#include "../../common/vector.h"

__global__ void add(double *a, double *b, double *c, int dim) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < dim) {
    c[idx] = a[idx] + b[idx];
  }
}

int main(const int argc, const char **argv) {
  double *a, *b, *c;             // host copies of a, b, c
  double *dev_a, *dev_b, *dev_c; // device copies of a, b, c
  int size; // bytes for a, b, c
  int vectorDim; // vector dimension
  int gridSize;  // grid size
  int blockSize; // block size

  if (argc < 3) {
    fprintf(stderr, "Usage: %s vectorDim blockSize\n", argv[0]);
    exit(1);
  }

  vectorDim = atoi(argv[1]);
  blockSize = atoi(argv[2]);

  if (vectorDim < 1) {
    fprintf(stderr, "Error: vectorDim expected >= 1, got %d\n", vectorDim);
    exit(1);
  }

  if (blockSize < 1) {
    fprintf(stderr, "Error: blockSize expected >= 1, got %d\n", blockSize);
    exit(1);
  }

  size = vectorDim * sizeof(double);

  // allocate host copies of a, b, c
  HANDLE_NULL(a = (double*)malloc(size));
  HANDLE_NULL(b = (double*)malloc(size));
  HANDLE_NULL(c = (double*)malloc(size));

  // allocate device copies of a, b, c
  HANDLE_ERROR(hipMalloc((void**)&dev_a, size));
  HANDLE_ERROR(hipMalloc((void**)&dev_b, size));
  HANDLE_ERROR(hipMalloc((void**)&dev_c, size));

  // fill a, b with random data
  random_vector_double(a, vectorDim);
  random_vector_double(b, vectorDim);

  // copy inputs to device
  HANDLE_ERROR(hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice));

  // launch add() kernel
  gridSize = vectorDim / blockSize;
  if (gridSize * blockSize < vectorDim) {
    gridSize += 1;
  }
  add<<< gridSize, blockSize >>>(dev_a, dev_b, dev_c, vectorDim);

  // copy device result back to host copy of c
  HANDLE_ERROR(hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost));

  // test result
  double *d;
  HANDLE_NULL(d = (double*)malloc(size));
  vector_add_double(a, b, d, vectorDim);
  if (!vector_equals_double(c, d, vectorDim)) {
    fprintf(stderr, "Error\n");
  } else {
    printf("Correct\n");
  }

  // free host
  free(a);
  free(b);
  free(c);
  free(d);

  // free device
  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipFree(dev_b));
  HANDLE_ERROR(hipFree(dev_c));

  return 0;
}
