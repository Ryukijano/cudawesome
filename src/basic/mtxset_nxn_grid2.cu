#include "hip/hip_runtime.h"
/*
 * @Name: mtxset_nxn_grid2.cu
 * @Description: Sets the elements of an integer square matrix.
 * The matrix is viewed as a single block of memory.
 * Blocks and threads are viewed as a 2D grid.
 *
 * @Author: Giacomo Marciani <gmarciani@acm.org>
 * @Institution: University of Rome Tor Vergata
 */

#include <stdio.h>
#include <math.h>
#include <../common/error.h>

#define MATRIX_DIM 9

__global__ void set(int *a, int dim) {
  int iX = blockIdx.x * blockDim.x + threadIdx.x;
  int iY = blockIdx.y * blockDim.y + threadIdx.y;
  int idx = iY * dim + iX;

  a[idx] = idx;
}

int main( void ) {
  int *a;     // host copy of a
  int *dev_a; // device copy of a
  int size = MATRIX_DIM * MATRIX_DIM * sizeof(int); // bytes for a matrix of MATRIX_DIM x MATRIX_DIM integers

  // allocate host copy of a
  a = HANDLE_NULL((int*)malloc(size));

  // allocate device copy of a
  HANDLE_ERROR(hipMalloc((void**)&dev_a, size));

  dim3 gridDim, blockDim;
  blockDim.x = 3;
  blockDim.y = 3;
  gridDim.x = MATRIX_DIM / blockDim.x;
  gridDim.y = MATRIX_DIM / blockDim.y;

  // launch set() kernel
  set<<< gridDim, blockDim >>>(dev_a, MATRIX_DIM);

  // copy device result back to host copy of a
  HANDLE_ERROR(hipMemcpy(a, dev_a, size, hipMemcpyDeviceToHost));

  // print result
  for(int y = 0; y < MATRIX_DIM; y++) {
    for(int x = 0; x < MATRIX_DIM; x++) {
        printf("%d ", a[y * MATRIX_DIM + x])
    }
    printf("\n");
  }

  // free host
  free(a);

  // free device
  HANDLE_ERROR(hipFree(dev_a));

  return 0;
}
